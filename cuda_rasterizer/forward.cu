#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

/**
 * OpenGL Mathematics(glm)：是针对图形编程的数学库，用于OpenGL的开发，这个库基于C++的模板库。
 * 提供了各种数学功能和数据结构：
 * 向量（vec2, vec3, vec4）
 * 矩阵（mat2, mat3, mat4）
 * 四元数（quaterion）
 * 常见的数学函数（平移、旋转、缩放、透视投影等）
 */

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
/**
 * 根据一个3D高斯的球谐系数，计算 当前相机中心 看向 该高斯中心 方向的RGB颜色值，(3,)
 * @param idx   当前高斯的索引
 * @param deg   当前的球谐阶数
 * @param max_coeffs    每个高斯的球谐系数个数=16
 * @param means 所有高斯 中心的世界坐标
 * @param campos 当前相机中心的世界坐标
 * @param shs    所有高斯的 球谐系数
 * @param clamped   输出的 所有高斯 是否被裁剪的标志 数组，某位置为 True表示：该高斯在当前相机的观测角度下，其RGB值3个的某个值 < 0，在后续渲染中不考虑它
 */
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// 该函数基于zhang等人的论文"Differentiable Point-Based Radiance Fields for Efficient View Synthesis"中的代码实现
	glm::vec3 pos = means[idx];		// 当前高斯中心 的世界坐标
	glm::vec3 dir = pos - campos;	// 从 相机中心 指向 当前高斯中心的 单位向量
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;   // 获取当前高斯的球谐系数(16, 3)

    // 基函数(SH_C0、SH_C1等) * 系数(sh) = 最终的球谐函数

    // 计算当前高斯的0阶SH系数的颜色值，(3,)
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0) {
        // 当前的球谐阶数 > 0，则计算一阶SH系数的颜色值
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1) {
            // 当前的球谐阶数 > 1，则计算二阶SH系数的颜色值
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2) {
                // 当前的球谐阶数 > 2，则计算三阶SH系数的颜色值
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
    // 为结果颜色值加上一个偏移量，(3,)
	result += 0.5f;

    // 将RGB颜色值限制在正值范围内。如果计算的 当前相机看该高斯的RGB颜色值 < 0，则在 geomState的 clamped中记录其RGB对应的值为True
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);  // 返回值>0的RGB值
}


/**
 * 3D协方差矩阵 ==> 2D协方差矩阵:
 * 1. 世界坐标系到相机坐标：viewmatirx
 * 2. 视锥到立方体：雅克比矩阵
 * @param mean      该高斯中心 的世界坐标
 * @param focal_x   相机在焦x轴方向上的焦距，也是视锥体近平面的深度
 * @param focal_y
 * @param tan_fovx  tan(Fovx / 2)
 * @param tan_fovy
 * @param cov3D     该高斯的 世界坐标系下的 3D协方差矩阵
 * @param viewmatrix 观测变换矩阵，W2C
 * @return 像素坐标系下的协方差矩阵，维度为(2,2)，但只返回了上半角元素(3个)
 */
__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
	// The following models the steps outlined by equations 29 and 31 in "EWA Splatting" (Zwicker et al., 2002).
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	// 雅克比矩阵在一个点附近才满足使用线性变换 近似 非线性变换的条件，而高斯的中心位置就是这个点，所以先求得3D高斯在相机坐标系下的位置）
	// 计算高斯中心在相机坐标系中的位置（在视锥中的位置）
	float3 t = transformPoint4x3(mean, viewmatrix);

    // 定义x和y方向的视锥限制
	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;

	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	// 构建雅克比矩阵（投影变换中的将视椎体压成立方体）
	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	// W：世界坐标系 ==> 相机坐标系的旋转矩阵 的转置
	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

    // 协方差矩阵从3D变为2D的公式：V_2D = JW V_3D W^T J^T
	glm::mat3 T = W * J;
    // 3D协方差矩阵
	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	// 2D协方差矩阵 = (W^T * J)^T * Vrk^T (W^T * J) = (J^T * W) * Vrk^T * (W^T * J)
	// [sigma_x sigma_xy]
	// [sigma_xy sigma_y]
	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

    // 确保 协方差矩阵 正定，数值稳定性考虑
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}

/**
 * 计算该高斯的 3D协方差矩阵（从旋转和缩放计算，需注意旋转四元数的归一化）
 * @param scale 该高斯的 缩放因子
 * @param mod   缩放因子调整系数
 * @param rot   该高斯的 旋转四元数
 * @param cov3D 输出的 协方差矩阵
 */
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
	// 创建 缩放矩阵(3x3)
	glm::mat3 S = glm::mat3(1.0f);  // 初始化为一个3维的单位阵
	S[0][0] = mod * scale.x;    // 将缩放因子填入主对角线元素中
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// 将输入的四元数归一化 以正确表示 旋转（假设已经是单位四元数，因此不再进行额外的标准化）
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// 四元数 => 旋转矩阵(3x3)
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

    // 计算M矩阵，即缩放后的旋转矩阵
	glm::mat3 M = S * R;

	// 计算世界坐标系下的 协方差矩阵(3x3)：R^T S^T S R
	glm::mat3 Sigma = glm::transpose(M) * M;

	// 因为协方差矩阵是对阵矩阵，因此只需存储上半角元素
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}


//! 为每个3D高斯进行预处理的CUDA核函数
// 计算投影圆圈的半径：在3D空间中的高斯分布投影到2D图像平面时，它通常会形成一个圆圈（实际上是椭圆，因为视角的影响）。这个步骤涉及计算这个圆圈的半径。
// 计算圆圈覆盖的像素数：这涉及到将图像平面分成许多小块（tiles），并计算每个高斯分布投影形成的圆圈与哪些小块相交。这是为了高效地渲染，只更新受影响的小块。
template<int C>
__global__ void preprocessCUDA(
    int P,  // 所有高斯的个数
    int D,  // 当前的球谐阶数
    int M,  // 每个高斯的球谐系数个数=16
	const float* orig_points,   // 所有高斯 中心的世界坐标 数组，(x0, y0, z0, ..., xn, yn, zn)
	const glm::vec3* scales,    // 所有高斯的 缩放因子
	const float scale_modifier, // 缩放因子的调整系数
	const glm::vec4* rotations, // 所有高斯的 旋转四元数
	const float* opacities,     // 所有高斯的 不透明度
	const float* shs,           // 所有高斯的 球谐系数
	bool* clamped,              // 输出的 所有高斯 是否被裁剪的标志 数组，某位置为 True表示：该高斯在当前相机的观测角度下，其RGB值3个的某个值 < 0，在后续渲染中不考虑它
	const float* cov3D_precomp, // 因预计算的3D协方差矩阵默认是空tensor，则传入的是一个 NULL指针
	const float* colors_precomp,    // 因预计算的颜色默认是空tensor，则传入的是一个 NULL指针
	const float* viewmatrix,    // 观测变换矩阵，W2C
	const float* projmatrix,    // 观测变换矩阵 * 投影变换矩阵，W2NDC = W2C * C2NDC
	const glm::vec3* cam_pos,   // 当前相机中心的世界坐标
	const int W, int H,         // 输出图像的宽、高
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	int* radii,                 // 输出的 所有高斯 投影在当前相机图像平面的最大半径 数组
	float2* points_xy_image,    // 输出的 所有高斯 中心在当前相机图像平面的二维坐标 数组
	float* depths,              // 输出的 所有高斯 中心在当前相机坐标系下的z值 数组
	float* cov3Ds,              // 输出的 所有高斯 在世界坐标系下的3D协方差矩阵 数组
	float* rgb,                 // 输出的 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组
	float4* conic_opacity,      // 输出的 所有高斯 2D协方差的逆 和 不透明度 数组
	const dim3 grid,            // CUDA网格的维度，grid.x是网格在x方向上的线程块数，grid.y是网格在y方向上的线程块数
	uint32_t* tiles_touched,    // 输出的 所有高斯 在当前相机图像平面覆盖的线程块 tile的个数 数组
	bool prefiltered)           // 预滤除的标志，默认为False
{
    // 1. 获取当前线程在CUDA grid中的全局索引，即当前线程处理的高斯的索引
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)   // 一个线程只处理一个高斯，避免越界访问
		return;

    // 2. 初始化该高斯 在图像平面的最大投影半径、覆盖的tile数量为 0。如果这些值保持为0，说明该高斯不会影响最终渲染，不需要进一步处理
	radii[idx] = 0;
	tiles_touched[idx] = 0;

    // 3. 检查该高斯是否在当前相机的视锥体内
	float3 p_view;  // 计算的 该高斯中心在相机坐标系下的三维坐标
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
        // 如果该高斯不在视锥体内，则直接返回
		return;

    // 4. 将该高斯投影到2D图像平面上
	// 4.1 将高斯的 中心 从3D变换到2D：包含观测变换、投影变换、视口变换、光栅化
    // (1) 该高斯中心的世界坐标
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	// (2) 变换该中心坐标：世界坐标系 ==> NDC坐标系（4维齐次坐标）
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);   // projmatrix = 观测变换 * 投影变换
	float p_w = 1.0f / (p_hom.w + 0.0000001f);  // 齐次坐标的归一化因子
    // (3) 映射到范围为[-1,1]的正方体中的三维坐标，用于后续2D投影
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

    // 4.2 将该高斯的 协方差矩阵 从3D变换到2D：包含观测变换(viewmatrix)、投影变换中的视锥到立方体的变换(雅可比矩阵)
	// (1) 获取世界坐标系下的 3D协方差矩阵
	const float* cov3D;
	if (cov3D_precomp != nullptr) {
		// 如果提供了预计算的3D协方差矩阵，则直接使用它
		cov3D = cov3D_precomp + idx * 6;
	} else {
		// 默认未提供，则从缩放因子和旋转四元数中计算世界坐标系下的 3D协方差矩阵，并存储在conv3Ds数组中
		computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}
    // (2) 变换该3D协方差矩阵：世界坐标系 ==观测变换==> 相机坐标系 ==投影变换中视锥到立方体==> 立方体中
    // 2D协方差矩阵（只存了上半角元素，3个）
	float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

    // 5. 计算2D协方差矩阵的 逆，用于EWA滤波算法
	float det = (cov.x * cov.z - cov.y * cov.y);    // 2x2方阵 xyyz的行列式 = xz - y^2
	if (det == 0.0f)
        // 行列式为0，该矩阵 不可逆，则直接返回
		return;
	float det_inv = 1.f / det;
    // 2D协方差矩阵的逆（也只存了上半角元素，2x2的矩阵的取逆是 主对角线对换，次对角线取负，再除以行列式）
	float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };  // 取逆

    // 6. 计算该3D高斯投影在屏幕平面上的、扩展后的 投影圆所在的矩形框边界，最后再转换为 线程块的坐标。如果矩形覆盖0个tile，则退出
    // (1) 计算该高斯 投影在图像平面的最大半径
	float mid = 0.5f * (cov.x + cov.z); // 2D协方差矩阵主对角线元素的均值
	float lambda1 = mid + sqrt(max(0.1f, mid * mid - det)); // 2D协方差矩阵的特征值，即代表2D椭圆的长轴和短轴
	float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));  // 投影在图像平面的最大半径 = 最长轴的3倍，覆盖99%的区域

    // (2) 计算该高斯 中心在图像平面的二维坐标（从 NDC平面 拉回到 图像平面）
    float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };

    // (3) 计算该高斯投影最大半径画的圆 在图像平面的影响范围（左上角和右下角坐标）对应在CUDA线程块（投影矩形）的边界
	uint2 rect_min, rect_max;
	getRect(point_image, my_radius, rect_min, rect_max, grid);

    if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
        // 投影矩形面积=0，说明该高斯不会影响任何屏幕像素，则直接返回
		return;

    // 7. 如果提供了预计算的颜色，则直接使用
	if (colors_precomp == nullptr) {
        // 默认，未预计算颜色，则根据 该高斯的球谐系数 与 当前相机看该高斯的方向 计算该观测下的RGB颜色值，(3,)，同时如果某个RGB值<0，则在 clamped数组对应位置中置为 True
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;    // C为模版参数，表示通道数，这里是3
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

    // 8. 存储计算的深度、半径、屏幕坐标等结果，用于下一步继续处理
	depths[idx] = p_view.z; // 该高斯中心在相机坐标系下的z值
	radii[idx] = my_radius; // 该高斯投影在图像平面的最大半径
	points_xy_image[idx] = point_image; // 该高斯中心在图像平面的二维坐标
	conic_opacity[idx] = { conic.x, conic.y, conic.z, opacities[idx] };     // 该高斯的2D协方差矩阵的逆、不透明度
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x); // 该高斯投影最大半径画的圆 在屏幕空间覆盖的tile数量，用于渲染的优化
}


//! 渲染：在一个block上协作渲染一个tile内各像素的RGB颜色值，每个线程负责一个像素
// 每个线程在 读取数据(把数据从公用显存拉到 block自己的显存) 和 进行计算 之间来回切换，使得线程们可以共同读取高斯数据，这样做的原因是block共享显存比公共显存快得多
template <uint32_t CHANNELS>    // CHANNELS = 3，即RGB三个通道
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)    // CUDA 启动核函数时使用的线程格和线程块的数量
renderCUDA(
	const uint2* __restrict__ ranges,   // 每个tile在 排序后的keys列表中的 起始和终止位置。索引：tile ID，值[x,y)：该tile在keys列表中起始、终止位置，个数y-x：落在该tile_ID上的高斯的个数。也可以用[x,y)在排序后的values列表中索引到该tile触及的所有高斯ID
	const uint32_t* __restrict__ point_list,    // 按 tile ID、高斯深度 排序后的 values列表，即 高斯ID 列表
	int W, int H,
	const float2* __restrict__ points_xy_image, // 所有高斯 中心在当前相机图像平面的二维坐标 的数组
	const float* __restrict__ features,         // 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组
	const float4* __restrict__ conic_opacity,   // 所有高斯 2D协方差矩阵的逆 和 不透明度 的数组
	float* __restrict__ final_T,                // 输出的 渲染后每个像素 pixel的 累积的透射率 的数组
	uint32_t* __restrict__ n_contrib,           // 输出的 渲染每个像素 pixel穿过的高斯的个数，也是最后一个对渲染该像素RGB值 有贡献的高斯ID 的数组
	const float* __restrict__ bg_color,         // 提供的背景颜色，默认为[1,1,1]，黑色
	float* __restrict__ out_color)              // 输出的 RGB图像（加上了背景颜色）
{
    // 1. 确定当前block处理的 tile的像素范围
    // pix_min： 当前处理的 tile的 左上角像素坐标
    // pix_max： 当前处理的 tile的 右下角像素坐标
    // pix：     当前处理的 像素 在像素平面的坐标
	auto block = cg::this_thread_block();   // 获取当前线程所处的 block（对应一个 tile）

    uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X; // 在水平方向上有多少个 block

    uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };       // block.group_index()：当前线程所处的 block在 grid中的三维索引
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
    uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };     // block.thread_index()：当前线程在 block中的三维索引

    uint32_t pix_id = W * pix.y + pix.x;        // 当前处理的 像素 在像素平面的　索引
	float2 pixf = { (float)pix.x, (float)pix.y };   // 当前处理的 像素 在像素平面的坐标

    // 2. 判断当前线程处理的 像素 是否在图像有效像素范围内
	bool inside = pix.x < W　&& pix.y < H;
    // 如果不在，则将 done设为 true，表示该线程不执行渲染操作
	bool done = !inside;

    // 3. 计算当前tile触及的高斯个数，太多，则分rounds批渲染
    // 根据当前处理的 tile_ID，获取该tile在排序后的keys列表中的起始、终止位置，[x,y)。个数y-x：投影到该tile上的高斯的个数。
    // 也可以用[x,y)在排序后的values列表中索引到该tile触及的所有高斯ID
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE); // 高斯个数过多，则分批处理，每批最多处理 BLOCK_SIZE=16*16个高斯
	int toDo = range.y - range.x;   // 当前tile还未处理的 高斯的个数

    // 4. 初始化同一block中的各线程共享的显存，分别定义三个共享显存数组，用于在每个block内共享数据
	__shared__ int collected_id[BLOCK_SIZE];        // 记录各线程处理的 高斯的ID
	__shared__ float2 collected_xy[BLOCK_SIZE];     // 记录各线程处理的高斯 中心在2D平面的 像素坐标
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];  // 记录各线程处理的高斯的2D协方差矩阵的 逆 和 不透明度

    // 5. 初始化渲染相关变量，包括当前像素颜色 C、贡献者数量
	float T = 1.0f;     // 透射率：光线经过高斯后 剩余的能量。初值设为 1
	uint32_t contributor = 0;       // 计算该像素经过了多少个高斯，也是最后一个对渲染当前像素RGB值 有贡献的高斯ID
	uint32_t last_contributor = 0;  // 存储最终经过的高斯球数量
	float C[CHANNELS] = { 0 };      // 最后渲染的颜色

    // 6. 外循环：迭代分批渲染任务，每批最多处理 BLOCK_SIZE = 16*16个高斯
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE) {
        // 检查该block内所有线程都已经完成渲染，则退出循环
		int num_done = __syncthreads_count(done);   // 通过 __syncthreads_count 函数统计当前block内 done变为 true的线程个数，如果全部线程都完成，则跳出循环
		if (num_done == BLOCK_SIZE)
			break;

        // 从全局显存中读取 每个高斯的数据到 当前block的共享显存（已经结束的线程去取）
		int progress = i * BLOCK_SIZE + block.thread_rank();    // 当前处理的线程ID。block.thread_rank()：当前线程在该 block内的ID，区间为[0, 线程数)
        // 当前线程ID有效，即其处理的高斯不越界
		if (range.x + progress < range.y)
		{
            // 当前线程处理的高斯的ID
			int coll_id = point_list[range.x + progress];

            collected_id[block.thread_rank()] = coll_id;    // 当前线程处理的高斯ID
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];   // 当前线程处理的高斯 中心在当前相机图像平面的像素坐标
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];  // 当前线程处理的高斯 2D协方差矩阵的逆 和 不透明度
		}
		block.sync();   // 迭代每个高斯后，同步当前block下的所有线程


        // 内循环：每个线程遍历当前block处理的tile触及的 当前批次的 所有高斯，进行基于锥体参数的渲染计算，并更新颜色信息
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++) {

			contributor++;  // 对渲染当前像素RGB值有贡献的高斯的个数

			// Resample using conic matrix (cf. "Surface Splatting" by Zwicker et al., 2001)
            // 计算当前高斯中心投影到像素平面的坐标 xy 与 当前像素的坐标 pixf 的差值，并使用锥体参数计算 power。
			float2 xy = collected_xy[j];    // 当前处理的2D高斯 中心的像素坐标
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };    // 当前处理像素 到 2D高斯中心像素坐标的 位移向量
			float4 con_o = collected_conic_opacity[j];          // 当前处理的高斯的 2D协方差矩阵的逆 和 不透明度，x、y、z: 分别是2D协方差逆矩阵的上半对角元素, w：不透明度

            // 2D高斯分布的指数部分，-1/2 d^T Σ^-1 d，用于确定像素在光栅化过程中的贡献程度
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

            // 当前高斯最终的不透明度（3DGS论文公式(2)中的α值，对光线的吸收程度）= 高斯椭球的不透明度 * 强度
			float alpha = min(0.99f, con_o.w * exp(power));

            if (alpha < 1.0f / 255.0f)
                // α太小，就将该高斯当作透明的
				continue;

            // 计算经过当前高斯后的 透射率（光线剩余的能量）= 累积经过之前高斯的 透射率 和 当前高斯的不透明度
            float test_T = T * (1 - alpha);
            // 透射率 < 极小值，光线能量太低，标记这个像素的渲染结束，不进行后续渲染
			if (test_T < 0.0001f) {
				done = true;
				continue;
			}

            // 通过 a-blending 计算当前像素的RGB三通道 颜色值 C。3DGS论文公式(3)
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T; // 每个通道的值是 累加 当前像素触及到的高斯 在当前相机中心的观测方向下 的RGB值 * a * 透射率T

			T = test_T;

            // 记录 渲染当前像素射线 穿过的高斯的个数，也是最后一个对渲染当前像素RGB值 有贡献的高斯ID
			last_contributor = contributor;
		}
	}
    // 当前像素渲染完成

    // 7. 写入最终渲染结果
	if (inside) {
        // 所有处理有效像素的 thread都会将其最终的渲染数据 写入帧缓冲区和辅助缓冲区

        final_T[pix_id] = T;    // 输出的 渲染像素pix_id的颜色过程中 累积的透射率
		n_contrib[pix_id] = last_contributor;   // 输出的 渲染像素pix_id的颜色过程中 穿过的高斯的个数，也是最后一个对渲染当前像素RGB值 最后一个有贡献的高斯ID

        // 最后输出的RGB颜色值 加上 背景颜色
        for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
	}
}

//! 渲染
void FORWARD::render(
	const dim3 grid,    // 定义的CUDA网格的维度，grid.x是网格在x方向上的线程块数，grid.y是网格在y方向上的线程块数，(W/16，H/16)
    dim3 block,         // 定义的线程块 block的维度，(16, 16, 1)
	const uint2* ranges,        // 每个tile在 排序后的keys列表中的 起始和终止位置。索引：tile ID，值[x,y)：该tile在keys列表中起始、终止位置，个数y-x：落在该tile_ID上的高斯的个数。也可以用[x,y)在排序后的values列表中索引到该tile触及的所有高斯ID
	const uint32_t* point_list, // 按 tile ID、高斯深度 排序后的 values列表，即 高斯ID 列表
	int W, int H,
	const float2* means2D,  // 已计算的 所有高斯 中心在当前相机图像平面的二维坐标
	const float* colors,    // 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组
	const float4* conic_opacity,    // 已计算的 所有高斯 2D协方差矩阵的逆 和 不透明度
	float* final_T,         // 输出的 渲染后每个像素 pixel的 累积的透射率 的数组
	uint32_t* n_contrib,    // 输出的 渲染每个像素 pixel穿过的高斯的个数，也是最后一个对渲染该像素RGB值 有贡献的高斯ID 的数组
	const float* bg_color,  // 背景颜色，默认为[1,1,1]，黑色
	float* out_color)       // 输出的 RGB图像（加上了背景颜色）
{
    // 开始进入CUDA并行计算，将图像分为多个线程块（分配一个 进程）；每个线程块为每个像素分配一个线程；
    // 对于每个block只排序一次，认为block里面的pixel都被block中的所有gaussian影响且顺序一样。
    // 在forward中，沿camera从前往后遍历gaussian，计算颜色累计值和透明度累计值，直到透明度累计超过1或者遍历完成，然后用背景色和颜色累计值和透明度累计值计算这个pixel的最终颜色。
    // 在backward中，遍历顺序与forward相反，从（之前记录下来的）最终透明度累计值和其对应的最后一个gaussian开始，从后往前算梯度。
	renderCUDA<NUM_CHANNELS> << <grid, block >> > (
		ranges,
		point_list,
		W, H,
		means2D,
		colors,
		conic_opacity,
		final_T,
		n_contrib,
		bg_color,
		out_color);
}

/**
 * 调用CUDA核函数 preprocessCUDA对每个高斯进行预处理和投影
 * 1. 将每个高斯投影到图像平面上，计算2D协方差矩阵、投影半径 radii；
 * 2. 计算投影所占的tile块坐标和个数 tile tiles_touched；
 * 3. 如果用球谐系数，将其转换成RGB；
 * 4. 记录高斯的像素坐标 points_xy_image
 */
void FORWARD::preprocess(
    int P,      // 所有高斯的个数
    int D,      // 当前的球谐阶数
    int M,      // 每个高斯的球谐系数个数=16
	const float* means3D,   // 所有高斯 中心的世界坐标 数组，(x0, y0, z0, ..., xn, yn, zn)
	const glm::vec3* scales,    // 所有高斯的 缩放因子
	const float scale_modifier, // 缩放因子的调整系数
	const glm::vec4* rotations, // 所有高斯的 旋转四元数
	const float* opacities,     // 所有高斯的 不透明度
	const float* shs,           // 所有高斯的 球谐系数
	bool* clamped,              // 输出的 所有高斯 是否被裁剪的标志 数组，某位置为 True表示：该高斯在当前相机的观测角度下，其RGB值3个的某个值 < 0，在后续渲染中不考虑它
	const float* cov3D_precomp, // 因预计算的3D协方差矩阵默认是空tensor，则传入的是一个 NULL指针
	const float* colors_precomp,    // 因预计算的颜色默认是空tensor，则传入的是一个 NULL指针
	const float* viewmatrix,    // 观测变换矩阵，W2C
	const float* projmatrix,    // 观测变换矩阵 * 投影变换矩阵，W2NDC = W2C * C2NDC
	const glm::vec3* cam_pos,   // 当前相机中心的世界坐标
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,   // tan(fov_x/2)和tan(fov_y/2)
	int* radii,             // 输出的 所有高斯 投影在当前相机图像平面的最大半径 数组
	float2* means2D,        // 输出的 所有高斯 中心在当前相机图像平面的二维坐标 数组
	float* depths,          // 输出的 所有高斯 中心在当前相机坐标系下的z值 数组
	float* cov3Ds,          // 输出的 所有高斯 在世界坐标系下的3D协方差矩阵 数组
	float* rgb,             // 输出的 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组
	float4* conic_opacity,  // 输出的 所有高斯 2D协方差的逆 和 不透明度 数组
	const dim3 grid,        // CUDA网格的维度，grid.x是网格在x方向上的线程块数，grid.y是网格在y方向上的线程块数
	uint32_t* tiles_touched,    // 输出的 所有高斯 在当前相机图像平面覆盖的线程块 tile的个数 数组
	bool prefiltered)       // 预滤除的标志，默认为False
{
    /**
     * 核函数使用__global__修饰符声明。这表明该函数是一个核函数（只能在 GPU上执行，不能在 CPU上执行）
     * 调用核函数时，需要使用特殊语法 << <numBlocks, blockSize> >>(data) 来指定执行配置
     * 这个配置包括两个部分：
     *    numBlocks：指定了多少个块（block）组成网格（grid），整个网格代表了所有并行执行单元的集合；
     *    blockSize：每个块中有多少个线程。块内的线程可以共享数据并协作执行任务。
     *
     * 线程ID：在CUDA核函数中，每个线程都会被分配一个唯一的线程ID。这个ID用于区分同一个核函数中不同的执行线程，使得每个线程可以处理数据的不同部分。例如，在处理数组时，线程ID可以用来确定每个线程负责处理数组中的哪个元素。
     * 获取线程ID：线程ID可以通过核函数的内置变量threadIdx来获取。在一维配置中，threadIdx.x表示当前线程的ID。如果使用二维或三维的块配置，还可以使用threadIdx.y和threadIdx.z。
     * 分成 (P+255)/256个block，每个block 256个thread
     */
	preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		depths,
		cov3Ds,
		rgb,
		conic_opacity,
		grid,
		tiles_touched,
		prefiltered
		);
}