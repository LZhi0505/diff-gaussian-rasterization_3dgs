#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>      // CUDA的CUB库
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>      // GLM (OpenGL Mathematics)库

#include <hip/hip_cooperative_groups.h>     // CUDA 9引入的Cooperative Groups库
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

/**
 * 引用库的介绍
 * 1. cooperative_groups库（同步）
 * __syncthreads()函数提供了在一个 block内同步各线程的方法，但有时要同步 block内的一部分线程或者多个 block的线程，这时候就需要 Cooperative Groups库。这个库定义了划分和同步一组线程的方法
 * 在3DGS中方法仅以两种方式被调用：
 * (1) auto idx = cg::this_grid().thread_rank();    其中 cg::this_grid()返回一个 cg::grid_group实例，表示当前线程所处的 grid。它有一个方法 thread_rank()返回当前线程在该 grid中排第几
 * (2) auto block = cg::this_thread_block();    其中 cg::this_thread_block返回一个 cg::thread_block实例，表示当前线程所处的 block，用到的成员函数有：
 *      block.sync()：同步该 block中的所有线程（等价于__syncthreads()）
 *      block.thread_rank()：返回非负整数，表示当前线程在该 block中排第几
 *      block.group_index()：返回一个 cg::dim3实例，表示该 block在 grid中的三维索引
 *      block.thread_index()：返回一个 cg::dim3实例，表示当前线程在 block中的三维索引
 *
 * 2. CUB库（并行处理）
 * 针对不同的计算等级：线程、wap、block、device等设计了并行算法。例如，reduce函数有四个版本：ThreadReduce、WarpReduce、BlockReduce、DeviceReduce
 * diff-gaussian-rasterization模块调用了CUB库的两个函数：
 * (1) hipcub::DeviceScan::InclusiveSum    计算前缀和，'InclusiveSum'是从第一个元素 累加到 当前元素 的和
 * (2) hipcub::DeviceRadixSort::SortPairs  device级别的并行基数 升序排序
 *
 * 3. GLM库
 * 专为图形学设计的只有头文件的C++数学库
 * 3DGS只用到了 glm::vec3（三维向量）, glm::vec4（四维向量）, glm::mat3（3×3矩阵）, glm::dot（向量点积）
 */


/**
 * 计算 tile总数的 二进制数中的 最高有效位 MSB的位置（二分法），用于确定位操作的范围
 * @param n CUDA网格的 tile总数
 */
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;   // 初值设为 n位数的一半，4 * 4 = 16 bit
	uint32_t step = msb;    // 初始步长
	while (step > 1)
	{
		step /= 2;      // 步长缩小一半
		if (n >> msb)   // 如果 n右移 msb位后不为 0，说明最高有效位在更高的位置
			msb += step;
		else            // 如果 n右移 msb位后为 0，说明最高有效位在更低的位置
			msb -= step;
	}
	if (n >> msb)   // 确保 msb是最高有效位的实际位置
		msb++;
	return msb;
}


// 检查某个线程的高斯是否在当前相机的视锥体内，bool类型输出到 present数组内
__global__ void checkFrustum(
    int P,          // 所有高斯的个数
	const float* orig_points,   // 所有高斯 中心的世界坐标
	const float* viewmatrix,    // 观测变换矩阵，W2C
	const float* projmatrix,    // 观测变换*投影变换矩阵，W2NDC = W2C * C2NDC
	bool* present)      // 输出的 所有高斯是否被当前相机看见的标志
{
    // 获取当前线程处理的高斯的索引
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;  // 输出的 该高斯在相机坐标系下的位置
    // 检查，如果不在当前相机视锥体内，则为False
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}


/**
 * 为每个高斯覆盖的所有 tile生成用于排序的 key-value，以便在后续操作中按深度对高斯进行排序
 * key：     uint64_t，前32位，其该高斯覆盖的每个tile 的ID，后32位，该高斯的 depth
 * value:   该高斯的 ID
 */
__global__ void duplicateWithKeys(
	int P,      // 所有高斯的个数
	const float2* points_xy,    // 预处理计算的 所有高斯 中心在当前相机图像平面的二维坐标 数组
	const float* depths,        // 预处理计算的 所有高斯 中心在当前相机坐标系下的z值（深度） 数组
	const uint32_t* offsets,    // 所有高斯 覆盖的 tile个数的 前缀和 数组
	uint64_t* gaussian_keys_unsorted,   // 输出的 遍历所有高斯生成它们覆盖的tile的 且 未排序的 keys 列表
	uint32_t* gaussian_values_unsorted, // 输出的 遍历所有高斯生成它们覆盖的tile的 且 未排序的 values 列表
	int* radii,     // 预处理计算的 所有高斯 投影在当前相机图像平面的最大半径 数组
	dim3 grid)      // CUDA网格的维度，grid.x是网格在x方向上的线程块数，grid.y是网格在y方向上的线程块数
{
    // 获取当前线程处理的高斯的索引
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

    // 只有该3D高斯投影到当前相机的图像平面的最大半径 > 0，即当前相机看见了该高斯，才生成 key-value
	if (radii[idx] > 0)
	{
        // 该高斯 前面的那些高斯已经覆盖的 tile的总数，即前一个高斯覆盖的tile的终止位置，也是该高斯覆盖的tile的起始位置
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
        // 计算该高斯投影到当前相机图像平面的 覆盖区域的左上角和右下角 tile块坐标
		uint2 rect_min, rect_max;
		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

        // 遍历该高斯 覆盖的每个 tile，为其生成一个 key-value
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;  // tile在整幅图像的 ID
				key <<= 32;         // 高位存 tile ID
				key |= *((uint32_t*) & depths[idx]);      // 低位存 该3D高斯在当前相机坐标系下的 深度

                // 为该高斯覆盖的当前 tile 分配 key-value
                gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;      // tile数组中的偏移量
			}
		}
	}
}


/**
 * 通过遍历排序后的 point_list_keys 列表，为每个 tile 计算出它在整个 point_list_keys 列表中的起始和终止位置，并将这些位置存储到 ranges 数组中
 */
__global__ void identifyTileRanges(
        int L,      // 排序的 tile总个数，即所有高斯 投影到二维图像平面上覆盖的 tile的总个数
        uint64_t* point_list_keys,  // 根据tile ID和高斯深度排序后的 keys列表
        uint2* ranges)  // ranges[tile_ID].x 和 y 表示 第 tile_ID个 tile在排过序的keys列表中的起始和终止位置
{
    // 获取当前线程的索引
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

    // 读取 当前线程处理的 key，[tile ID | 深度]
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;  // 当前tile的ID
	if (idx == 0)
        // 如果是第一个 tile，则其起始位置在索引 0 处
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
        // 当前tile 和 前一个tile 不同，则记录前一个tile的终止位置和当前tile的起始位置
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;   // 前一个 tile的终止位置是当前索引 idx
			ranges[currtile].x = idx;   // 当前 tile的起始位置也是 idx
		}
	}
	if (idx == L - 1)
        // 如果是最后一个 tile，则其终止位置在索引 L 处
		ranges[currtile].y = L;
}

// 检查所有高斯是否被在当前相机的视锥体内，即是否被当前相机看见，标志保存在 present数组中
void CudaRasterizer::Rasterizer::markVisible(
	int P,          // 所有高斯的个数
	float* means3D,     // 所有高斯 中心的世界坐标
	float* viewmatrix,  // 观测变换矩阵，W2C
	float* projmatrix,  // 观测变换*投影变换矩阵，W2NDC = W2C * C2NDC
	bool* present)      // 输出的 所有高斯是否被当前相机看见的标志
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

// CUDA内存状态类，用于在GPU内存中存储和管理不同类型的数据
// fromChunk：从以 char数组形式存储的二进制块中读取 GeometryState、ImageState、BinningState等类的信息
/**
 * (1) 存储与高斯几何相关的信息，从动态分配的内存块(char*& chunk)中 提取并初始化 GeometryState结构（与高斯各参数的数据成员）
 * 使用 obtain 函数为 GeometryState 的不同成员分配空间，并返回一个初始化的 GeometryState 实例
 * @param chunk 一个指向内存块的指针引用
 * @param P     所有高斯的个数
 */
CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);         // 所有高斯 在相机坐标系下的深度
	obtain(chunk, geom.clamped, P * 3, 128);    // 所有高斯 是否被裁剪的标志
	obtain(chunk, geom.internal_radii, P, 128); // 所有高斯 在图像平面上的投影半径
	obtain(chunk, geom.means2D, P, 128);    // 输出的 所有高斯 中心投影到图像平面的坐标
	obtain(chunk, geom.cov3D, P * 6, 128);  // 所有高斯 在世界坐标系下的3D协方差矩阵
	obtain(chunk, geom.conic_opacity, P, 128);  // 所有高斯的 2D协方差的逆、不透明度
	obtain(chunk, geom.rgb, P * 3, 128);    // 所有高斯的 RGB颜色
	obtain(chunk, geom.tiles_touched, P, 128);  // 所有高斯 覆盖的 tile数量

    // 计算前缀和，InclusiveSum表示包括自身，ExclusiveSum表示不包括自身
    // 当临时所需的显存空间为 NULL时，所需的分配空间大小被写入到 第二个参数中，并且不执行任何操作
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);

    obtain(chunk, geom.scanning_space, geom.scan_size, 128);    // 用于计算前缀和的中间缓冲区，数据的对齐方式为 128字节
	obtain(chunk, geom.point_offsets, P, 128);  // 每个高斯在有序列表中的位置
	return geom;
}

// (2) 存储与图像渲染相关的信息
CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128); // 每个像素的累积 alpha值
	obtain(chunk, img.n_contrib, N, 128);   // 每个像素的贡献高斯数量
	obtain(chunk, img.ranges, N, 128);      // 每个 tile 所需的高斯范围
	return img;
}

/**
 * (3) 初始化 BinningState 实例，分配所需的内存，并执行排序操作
 */
CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);  // 排序后的高斯分布索引列表
	obtain(chunk, binning.point_list_unsorted, P, 128); // 未排序的高斯分布索引列表
	obtain(chunk, binning.point_list_keys, P, 128);     // 排序后的 (tile, depth) 键列表
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);    // 未排序的 (tile, depth) 键列表

    // GPU上device级别的并行基数 升序排序, 将 point_list_keys_unsorted 作为键，point_list_unsorted 作为值进行排序，排序结果存储在 point_list_keys 和 point_list 中
    hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);

    // list_sorting_space 用于排序的临时空间
    obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}


/**
 * 高斯的可微光栅化的前向渲染处理，可当作 main 函数
 */
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,   // 三个都是调整内存缓冲区的函数指针
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P,    // 所有高斯的个数
    int D,      // 当前的球谐阶数
    int M,      // 每个高斯的球谐系数个数=16
	const float* background,    // 背景颜色，默认为[1,1,1]，黑色
	const int width, int height,    // 图像宽、高
	const float* means3D,   // 所有高斯 中心的世界坐标
	const float* shs,       // 所有高斯的 球谐系数
	const float* colors_precomp,    // 因预计算的颜色默认是空tensor，则其传入的是一个 NULL指针
	const float* opacities, // 所有高斯的 不透明度
	const float* scales,    // 所有高斯的 缩放因子
	const float scale_modifier, // 缩放因子的调整系数
	const float* rotations,     // 所有高斯的 旋转四元数
	const float* cov3D_precomp, // 因预计算的3D协方差矩阵默认是空tensor，则其传入的是一个 NULL指针
	const float* viewmatrix,    // 观测变换矩阵，W2C
	const float* projmatrix,    // 观测变换矩阵 * 投影变换矩阵，W2NDC = W2C * C2NDC
	const float* cam_pos,       // 当前相机中心的世界坐标
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,     // 预滤除的标志，默认为False
	float* out_color,       // 输出的 颜色图像，(3,H,W)
	int* radii,             // 输出的 在图像平面上的投影半径(N,)
	bool debug)     // 默认为False
{
    // 1. 计算焦距，W = 2fx * tan(Fovx/2) ==> fx = W / (2 * tan(Fovx/2))
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

    // 2. 分配、初始化几何信息 geomState
	size_t chunk_size = required<GeometryState>(P);     // 根据高斯的数量 P，计算存储所有高斯各参数 所需的空间大小
	char* chunkptr = geometryBuffer(chunk_size);        // 分配指定大小 chunk_size的缓冲区，即给所有高斯的各参数分配存储空间，返回指向该存储空间的指针 chunkptr
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);    // 在给定的内存块中初始化 GeometryState 结构体, 为不同成员分配空间，并返回一个初始化的实例

	if (radii == nullptr) {
        // 如果传入的、要输出的 高斯在图像平面的投影半径为 nullptr，则将其设为
		radii = geomState.internal_radii;
	}

    // 3. 定义一个 tile_grid的维度，即在水平和垂直方向上需要多少个线程块来覆盖整个渲染区域，(W/16，H/16)
	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
    // 定义一个 block的维度，即在水平和垂直方向上的线程数。每个线程处理一个像素，则每个线程块处理16*16个像素，(16, 16, 1)
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// 4. 分配、初始化图像信息 ImageState
	size_t img_chunk_size = required<ImageState>(width * height);   // 计算存储所有2D像素各参数 所需的空间大小
	char* img_chunkptr = imageBuffer(img_chunk_size);                  // 分配存储空间, 并返回指向该存储空间的指针
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);  // 在给定的内存块中初始化 ImageState 结构体, 为不同成员分配空间，并返回一个初始化的实例

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr) {
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

    //! 5. 预处理和投影：将每个高斯投影到图像平面上、计算投影所占的tile块坐标和个数、根据球谐系数计算RGB值。 具体实现在 forward.cu/preprocessCUDA
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,      // geomState中记录高斯是否被裁剪的标志，即某位置为 True表示：该高斯在当前相机的观测角度下，其RGB值3个的某个值 < 0，在后续渲染中不考虑它
		cov3D_precomp,          // 因预计算的3D协方差矩阵默认是空tensor，则传入的是一个 NULL指针
		colors_precomp,         // 因预计算的颜色默认是空tensor，则传入的是一个 NULL指针
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,              // 输出的 所有高斯 投影在当前相机图像平面的最大半径 数组
		geomState.means2D,  // 输出的 所有高斯 中心在当前相机图像平面的二维坐标 数组
		geomState.depths,   // 输出的 所有高斯 中心在当前相机坐标系下的z值 数组
		geomState.cov3D,    // 输出的 所有高斯 在世界坐标系下的3D协方差矩阵 数组
		geomState.rgb,      // 输出的 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组
		geomState.conic_opacity,    // 输出的 所有高斯 2D协方差的逆 和 不透明度 数组
		tile_grid,                  // CUDA网格的维度，grid.x是网格在x方向上的线程块数，grid.y是网格在y方向上的线程块数
		geomState.tiles_touched,    // 输出的 所有高斯 在当前相机图像平面覆盖的线程块 tile的个数 数组
		prefiltered                 // 预滤除的标志，默认为False
	), debug)

    //! 6. 高斯排序和合成顺序：根据高斯距离摄像机的远近来计算每个高斯在Alpha合成中的顺序
    // ---开始--- 通过视图变换 W 计算出像素与所有重叠高斯的距离，即这些高斯的深度，形成一个有序的高斯列表
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
    // 在GPU上并行计算 每个高斯投影到当前相机图像平面上 2D高斯覆盖的 tile个数的 前缀和，结果存储在 point_offsets，提供了每个高斯覆盖tile区域的累加结束位置
    // 是为 所有高斯投影到图像平面上覆盖的所有 tile分配唯一的 ID
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space,  // 额外需要的临时显存空间
                                             geomState.scan_size,       // 临时显存空间的大小
                                             geomState.tiles_touched,   // 输入指针，已计算的 每个高斯 投影到当前相机图像平面覆盖的 tile个数的 数组
                                             geomState.point_offsets,   // 输出指针，指向一个数组，每个元素是 从第一个高斯到当前高斯所覆盖的所有 tile的 数量
                                             P      // 所有高斯的个数
                                             ), debug)

    // 计算所有高斯 投影到二维图像平面上覆盖的 tile的总个数
	int num_rendered;
    // 将 point_offsets数组的最后一个元素，即所有高斯投影到当前相机图像平面上所覆盖的 tile的 总数，从GPU复制到CPU的变量 num_rendered中
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

    // 6.3 分配、初始化排序信息 BinningState，存储要 排序的 key-value对 和 排序后的结果
	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

    // 遍历每个高斯，记录其落在的 tile ID 与 深度
	// 对于每个要渲染的高斯, 为其覆盖的所有tile生成排序所用的 key-value，其中，key：[tile ID | 3D高斯的深度]；value：[对应3D高斯的 ID]
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,  // 预处理计算的 所有高斯 中心在当前相机图像平面的二维坐标 数组
		geomState.depths,   // 预处理计算的 所有高斯 中心在当前相机坐标系下的z值（深度） 数组
		geomState.point_offsets,    // 所有高斯覆盖的 tile个数的 前缀和
		binningState.point_list_keys_unsorted,  // 输出的 遍历所有高斯生成它们覆盖的tile的 且 未排序的 keys 列表（uint64_t）
		binningState.point_list_unsorted,       // 输出的 遍历所有高斯生成它们覆盖的tile的 且 未排序的 values 列表
		radii,          // 预处理计算的 所有高斯 投影在当前相机图像平面的最大半径 数组
		tile_grid)      // CUDA网格的维度，grid.x是网格在x方向上的线程块数，grid.y是网格在y方向上的线程块数
	CHECK_CUDA(, debug)

    // 计算 tile总数的 二进制数中的 最高有效位的 位置，用于确定位操作的范围
	int bit = getHigherMsb(tile_grid.x * tile_grid.y);


    // 遍历每个 tile，根据落在其上的高斯的深度 进行升序排序
    // 按 key的大小 对（keys [tile ID | 3D高斯的深度], values [高斯ID]）进行稳定的、并行、基数 升序排序：每个 tile对应的多个高斯按深度升序排放在一起
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,    // 排序时用到的临时显存空间
		binningState.sorting_size,                     // 临时显存空间的大小
		binningState.point_list_keys_unsorted,  // 未排序的 每个高斯覆盖的所有 tile的 keys列表，[tile ID | 3D高斯的深度]
        binningState.point_list_keys,           // 排序后的 keys列表
		binningState.point_list_unsorted,   // 未排序的 每个高斯覆盖的所有 tile的 values列表，[对应3D高斯的 ID]
        binningState.point_list,            // 排序后的 values列表
		num_rendered,   // 要排序的 tile总个数，即所有高斯 投影到二维图像平面上覆盖的 tile的总个数
        0,      // 指定时从最低位开始
        32 + bit    // 指定排序的最高位，表示排序的范围是从第 0位到第 32 + bit位。bit代表了 tile ID的最高位数。加上 32 是因为 tile ID和深度值分别占据了32位
        ), debug)

    // 将CUDA设备内存中的一块区域 imgState.ranges 数组中的所有元素初始化为 0（uint2是一个由两个 uint32_t组成的结构体，所以其大小是 8字节）
	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);


    // 根据排序后的keys列表，为每个 tile 计算 其在排序后的keys列表中的起始和终止位置，后续的渲染或处理步骤可以根据 tile ID 快速找到这个 tile 对应的高斯对象，而不需要再次进行复杂的查找或遍历
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,       // 排序的 tile总个数，即所有高斯 投影到二维图像平面上覆盖的 tile的总个数
			binningState.point_list_keys,   // 根据tile ID和高斯深度排序后的 keys列表
			imgState.ranges);   // 输出的 每个tile在 排序后的keys列表中的 起始和终止位置。索引：tile ID，值[x,y)：该tile在keys列表中起始、终止位置，个数表示多少个高斯落在该tile内
	CHECK_CUDA(, debug)


    // 每个tile并行地 blend涉及的高斯
    // 如果传入的预计算的颜色 不是空指针，则是预计算的颜色
    //                    是空指针，则是预处理中 计算的 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;

    //! 7. 渲染
    // 一个线程负责一个像素，一个block负责一个tile。线程在读取数据（把数据从公用显存拉到 block自己的显存）和进行计算之间来回切换，使得线程们可以共同读取高斯数据，这样做的原因是block共享内存比公共显存快得多。具体实现在 forward.cu/renderCUDA
	CHECK_CUDA(FORWARD::render(
		tile_grid,     // 定义的CUDA网格的维度，grid.x是网格在x方向上的线程块数，grid.y是网格在y方向上的线程块数，(W/16，H/16)
        block,              // 定义的线程块 block的维度，(16, 16, 1)
		imgState.ranges,    // 每个tile在 排序后的keys列表中的 起始和终止位置。索引：tile ID，值[x,y)：该tile在keys列表中起始、终止位置，个数表示多少个高斯落在该tile内
		binningState.point_list,    // 按 tile ID、高斯深度 排序后的 高斯ID 列表
		width, height,
		geomState.means2D,  // 已计算的 所有高斯 中心在当前相机图像平面的二维坐标 数组
		feature_ptr,        // 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组
		geomState.conic_opacity,    // 已计算的 所有高斯 2D协方差矩阵的逆 和 不透明度 数组
		imgState.accum_alpha,   // 输出的 渲染过程后 每个像素 pixel的最终透明度或透射率值
		imgState.n_contrib,     // 输出的 每个像素 pixel的最后一个贡献的高斯是谁
		background,     // 背景颜色，默认为[1,1,1]，黑色
		out_color               // 输出的 RGB图像
        ), debug)

	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding to forward render pass
// 产生对应于前向渲染过程所需的优化梯度
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
    // 根据每像素损失梯度计算损失梯度，关于2D均值位置、圆锥矩阵、
    // 高斯的不透明度和RGB。如果我们获得了预计算的颜色而不是球谐系数，就使用它们。
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
    //! 核心渲染函数，定义在backward.h中，具体实现在 backward.cu/renderCUDA
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor), debug)

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
    // 处理预处理的剩余部分
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot), debug)
}