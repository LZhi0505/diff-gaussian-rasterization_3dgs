#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>      // CUDA的CUB库
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>      // GLM (OpenGL Mathematics)库

#include <hip/hip_cooperative_groups.h>     // CUDA 9引入的Cooperative Groups库
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

/**
 * 引用库的介绍
 * 1. cooperative_groups库（同步）
 * __syncthreads()函数提供了在一个 block内同步各线程的方法，但有时要同步 block内的一部分线程或者多个 block的线程，这时候就需要 Cooperative Groups库。这个库定义了划分和同步一组线程的方法
 * 在3DGS中方法仅以两种方式被调用：
 * (1) auto idx = cg::this_grid().thread_rank();    其中 cg::this_grid()返回一个 cg::grid_group实例，表示当前线程所处的 grid。它有一个方法 thread_rank()返回当前线程在该 grid中排第几
 * (2) auto block = cg::this_thread_block();    其中 cg::this_thread_block返回一个 cg::thread_block实例，表示当前线程所处的 block，用到的成员函数有：
 *      block.sync()：同步该 block中的所有线程（等价于__syncthreads()）
 *      block.thread_rank()：返回非负整数，表示当前线程在该 block中排第几
 *      block.group_index()：返回一个 cg::dim3实例，表示该 block在 grid中的三维索引
 *      block.thread_index()：返回一个 cg::dim3实例，表示当前线程在 block中的三维索引
 *
 * 2. CUB库（并行处理）
 * 针对不同的计算等级：线程、wap、block、device等设计了并行算法。例如，reduce函数有四个版本：ThreadReduce、WarpReduce、BlockReduce、DeviceReduce
 * diff-gaussian-rasterization模块调用了CUB库的两个函数：
 * (1) hipcub::DeviceScan::InclusiveSum    计算前缀和，'InclusiveSum'是从第一个元素 累加到 当前元素 的和
 * (2) hipcub::DeviceRadixSort::SortPairs  device级别的并行基数 升序排序
 *
 * 3. GLM库
 * 专为图形学设计的只有头文件的C++数学库
 * 3DGS只用到了 glm::vec3（三维向量）, glm::vec4（四维向量）, glm::mat3（3×3矩阵）, glm::dot（向量点积）
 */


/**
 * 计算 tile总数的 二进制数中的 最高有效位 MSB的位置（二分法），用于确定位操作的范围
 * @param n CUDA网格的 tile总数
 */
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;   // 初值设为 n位数的一半，4 * 4 = 16 bit
	uint32_t step = msb;    // 初始步长
	while (step > 1)
	{
		step /= 2;      // 步长缩小一半
		if (n >> msb)   // 如果 n右移 msb位后不为 0，说明最高有效位在更高的位置
			msb += step;
		else            // 如果 n右移 msb位后为 0，说明最高有效位在更低的位置
			msb -= step;
	}
	if (n >> msb)   // 确保 msb是最高有效位的实际位置
		msb++;
	return msb;
}



/**
 * 检查某个线程处理的高斯是否在当前相机的视锥体内
 */
__global__ void checkFrustum(
    int P,          // 所有高斯的个数
	const float* orig_points,   // 所有高斯 中心的世界坐标
	const float* viewmatrix,    // 观测变换矩阵，W2C
	const float* projmatrix,    // 观测变换*投影变换矩阵，W2NDC = W2C * C2NDC
	bool* present)      // 输出的 所有高斯 是否在当前相机视锥体内的标志 数组
{
    // 获取当前线程处理的高斯的索引
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;  // 该高斯在当前相机坐标系下的坐标
    // 检查该高斯是否在当前相机的视锥体内
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}


/**
 * 为每个高斯覆盖的所有 tile生成用于排序的 key-value，以便在后续操作中按深度对高斯进行排序
 */
__global__ void duplicateWithKeys(
	int P,      // 所有高斯的个数
	const float2* points_xy,    // 预处理计算的 所有高斯 中心在当前相机图像平面的二维坐标 数组
	const float* depths,        // 预处理计算的 所有高斯 中心在当前相机坐标系下的z值（深度）数组
	const uint32_t* offsets,    // 所有高斯 覆盖的 tile个数的 前缀和 数组
	uint64_t* gaussian_keys_unsorted,   // 输出的 遍历所有高斯生成它们覆盖的tile的 且 未排序的 keys 列表。每个元素64bit，高32位存某高斯覆盖的tile_ID，低32位存某高斯中心在相机坐标系下的z（深度）值
	uint32_t* gaussian_values_unsorted, // 输出的 遍历所有高斯生成它们覆盖的tile的 且 未排序的 values 列表。每个元素是 某高斯的ID
	int* radii,     // 预处理计算的 所有高斯 投影在当前相机图像平面的最大半径 数组
	dim3 grid)      // CUDA网格的维度，grid.x是网格在x方向上的线程块数，grid.y是网格在y方向上的线程块数
{
    // 获取当前线程处理的高斯的索引
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

    // 只有该3D高斯投影到当前相机的图像平面的最大半径 > 0，即当前相机看见了该高斯，才生成 key-value
	if (radii[idx] > 0)
	{
        // 该高斯 前面的那些高斯已经覆盖的 tile的总数，即前一个高斯覆盖的tile的终止位置，也是该高斯覆盖的tile的起始位置
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
        // 计算该高斯投影到当前相机图像平面的 覆盖区域的左上角和右下角 tile块坐标
		uint2 rect_min, rect_max;
		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

        // 遍历该高斯 覆盖的每个 tile，为其生成一个 key-value：tile_ID|高斯深度 - 该高斯的ID
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;  // tile在整幅图像的 ID
				key <<= 32;         // 高位存 tile ID
				key |= *((uint32_t*) & depths[idx]);      // 低位存 该3D高斯在当前相机坐标系下的 深度

                // 为该高斯覆盖的当前 tile 分配 key-value
                gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;      // tile数组中的偏移量
			}
		}
	}
}


/**
 * 通过遍历排序后的 point_list_keys 列表，为每个 tile 计算出它在整个 point_list_keys 列表中的起始和终止位置，并将这些位置存储到 ranges 数组中
 */
__global__ void identifyTileRanges(
        int L,      // 排序的 tile总个数，即所有高斯 投影到二维图像平面上覆盖的 tile的总个数
        uint64_t* point_list_keys,  // 根据tile ID和高斯深度排序后的 keys列表
        uint2* ranges)  // ranges[tile_ID].x 和 y 表示 第 tile_ID个 tile在排过序的keys列表中的起始和终止位置
{
    // 获取当前线程的索引
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

    // 读取 当前线程处理的 key，[tile ID | 深度]
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;  // 当前tile的ID
	if (idx == 0)
        // 如果是第一个 tile，则其起始位置在索引 0 处
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
        // 当前tile 和 前一个tile 不同，则记录前一个tile的终止位置和当前tile的起始位置
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;   // 前一个 tile的终止位置是当前索引 idx
			ranges[currtile].x = idx;   // 当前 tile的起始位置也是 idx
		}
	}
	if (idx == L - 1)
        // 如果是最后一个 tile，则其终止位置在索引 L 处
		ranges[currtile].y = L;
}


/**
 * 检查所有高斯是否在当前相机的视锥体内
 */
void CudaRasterizer::Rasterizer::markVisible(
	int P,          // 所有高斯的个数
	float* means3D,     // 所有高斯 中心的世界坐标
	float* viewmatrix,  // 观测变换矩阵，W2C
	float* projmatrix,  // 观测变换*投影变换矩阵，W2NDC = W2C * C2NDC
	bool* present)      // 输出的 所有高斯 是否在当前相机视锥体内的标志 数组
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

// CUDA内存状态类，用于在GPU内存中存储和管理不同类型的数据
// fromChunk：从以 char数组形式存储的二进制块中读取 GeometryState、ImageState、BinningState等类的信息
/**
 * (1) 存储与高斯几何相关的信息，从动态分配的内存块(char*& chunk)中 提取并初始化 GeometryState结构（与高斯各参数的数据成员）
 * 使用 obtain 函数为 GeometryState 的不同成员分配空间，并返回一个初始化的 GeometryState 实例
 * @param chunk 一个指向内存块的指针引用
 * @param P     所有高斯的个数
 */
CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);         // 所有高斯 中心在当前相机坐标系下的z值（深度） 数组
	obtain(chunk, geom.clamped, P * 3, 128);    // 所有高斯 是否被裁剪的标志 数组
	obtain(chunk, geom.internal_radii, P, 128); // 所有高斯 在图像平面上的投影半径
	obtain(chunk, geom.means2D, P, 128);        // 所有高斯 中心投影在当前相机图像平面的二维坐标 数组
	obtain(chunk, geom.cov3D, P * 6, 128);  // 所有高斯 在世界坐标系下的3D协方差矩阵 数组
	obtain(chunk, geom.conic_opacity, P, 128);  // 所有高斯 2D协方差的逆 和 不透明度 数组
	obtain(chunk, geom.rgb, P * 3, 128);    // 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组
	obtain(chunk, geom.tiles_touched, P, 128);  // 所有高斯 在当前相机图像平面覆盖的线程块 tile的个数 数组

    // 计算前缀和，InclusiveSum表示包括自身，ExclusiveSum表示不包括自身
    // 当临时所需的显存空间为 NULL时，所需的分配空间大小被写入到 第二个参数中，并且不执行任何操作
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);

    obtain(chunk, geom.scanning_space, geom.scan_size, 128);    // 用于计算前缀和的中间缓冲区，数据的对齐方式为 128字节
	obtain(chunk, geom.point_offsets, P, 128);  // 所有高斯 覆盖的 tile个数的 前缀和 数组，每个元素是 从第一个高斯到当前高斯所覆盖的所有tile的数量
	return geom;
}

/**
 * (2) 给 ImageState img分配所需的内存
 * @param N 图片中 tile的总数
 */
CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128); // 渲染后每个像素 pixel的 累积的透射率 的数组
	obtain(chunk, img.n_contrib, N, 128);   // 渲染每个像素 pixel穿过的高斯的个数，也是最后一个对渲染该像素RGB值 有贡献的高斯ID 的数组
	obtain(chunk, img.ranges, N, 128);      // 每个tile在 排序后的keys列表中的 起始和终止位置。索引：tile_ID；值[x,y)：该tile在keys列表中起始、终止位置，个数y-x：落在该tile_ID上的高斯的个数
	return img;
}

/**
 * (3) 给 BinningState binning分配所需的内存，并执行排序操作
 */
CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);  // 排序后的 value列表
	obtain(chunk, binning.point_list_unsorted, P, 128); // 未排序的 所有高斯覆盖的tile的 values列表，每个元素是 对应高斯的ID

    obtain(chunk, binning.point_list_keys, P, 128);     // 排序后的 keys列表，分布顺序：大顺序：各tile_ID，小顺序：落在该tile内各高斯的深度
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);    // 未排序的 所有高斯覆盖的tile的 keys列表，分布顺序：大顺序：各高斯，小顺序：该高斯覆盖的各tile_ID。每个元素是 (tile_ID | 3D高斯的深度)

    // GPU上device级别的并行基数 升序排序, 将 point_list_keys_unsorted 作为键，point_list_unsorted 作为值进行排序，排序结果存储在 point_list_keys 和 point_list 中
    hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);

    // list_sorting_space 用于排序的临时空间
    obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}


/**
 * 可微光栅化的 前向传播，可当作 main 函数
 * 1: 分配 (p+255/256)个 block，每个 block有 256个 thread，对每个高斯做 preprocessCUDA();
 * 2: 生成 buffer并对高斯做排序；
 * 3: 分配 num_tiles个 block，每个 block有 256个 thread，对每个 pixel做渲染
 * return：num_rendered：所有高斯 投影到二维图像平面上覆盖的 tile的总个数 = 排序列表的长度
 */
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,   // 三个都是调整内存缓冲区的函数指针
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P,    // 所有高斯的个数
    int D,      // 当前的球谐阶数
    int M,      // 每个高斯的球谐系数个数=16
	const float* background,    // 背景颜色，默认为[1,1,1]，黑色
	const int width, int height,    // 图像宽、高
	const float* means3D,   // 所有高斯 中心的世界坐标
	const float* shs,       // 所有高斯的 球谐系数
	const float* colors_precomp,    // 因预计算的颜色默认是空tensor，则其传入的是一个 NULL指针
	const float* opacities, // 所有高斯的 不透明度
	const float* scales,    // 所有高斯的 缩放因子
	const float scale_modifier, // 缩放因子的调整系数
	const float* rotations,     // 所有高斯的 旋转四元数
	const float* cov3D_precomp, // 因预计算的3D协方差矩阵默认是空tensor，则其传入的是一个 NULL指针
	const float* viewmatrix,    // 观测变换矩阵，W2C
	const float* projmatrix,    // 观测变换矩阵 * 投影变换矩阵，W2NDC = W2C * C2NDC
	const float* cam_pos,       // 当前相机中心的世界坐标
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,     // 预滤除的标志，默认为False
	float* out_color,       // 输出的 RGB图像，考虑了背景颜色，(3,H,W)
	int* radii,             // 输出的 所有高斯 投影在当前相机图像平面的最大半径 数组，(N,)
	bool debug)     // 默认为False
{
    // 1. 计算焦距，W = 2fx * tan(Fovx/2) ==> fx = W / (2 * tan(Fovx/2))
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

    // 2. 为 GeometryState分配显存，每个高斯都有一个 GeometryState数据
	size_t chunk_size = required<GeometryState>(P);     // 根据高斯的数量 P，模版函数 required调用 fromChunk函数来获取内存，返回结束地址，也即所需的存储空间大小
	char* chunkptr = geometryBuffer(chunk_size);        // 根据所需的存储空间大小，调用 rasterize_points.cu文件中的 resizeFunctional函数里面嵌套的匿名函数 lambda来调整显存块大小，并返回首地址 chunkptr
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);    // 用显存块首地址作为参数，调用 fromChunk函数为 GeometryState geo申请显存

	if (radii == nullptr) {
        // 如果传入的、要输出的 高斯在图像平面的投影半径 为空指针，则将其设为 geomState缓存的投影半径
		radii = geomState.internal_radii;
	}

    // 3. 定义一个 tile_grid的维度，即在水平和垂直方向上需要多少个线程块 block来覆盖整个渲染区域，(W/16, H/16, 1)
	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
    // 定义一个 block的维度，即在水平和垂直方向上的线程 thread个数。每个线程处理一个像素，则每个block处理16*16个像素，(16, 16, 1)
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// 4. 为 ImageState分配显存，每个像素都有一个 ImageState数据
	size_t img_chunk_size = required<ImageState>(width * height);   // 计算存储所有2D像素各参数 所需的空间大小
	char* img_chunkptr = imageBuffer(img_chunk_size);                  // 分配存储空间, 并返回指向该存储空间的指针
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);  // 在给定的内存块中初始化 ImageState 结构体, 为不同成员分配空间，并返回一个初始化的实例

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr) {
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

    //! 5. 预处理和投影。具体实现在 forward.cu/preprocessCUDA
    // (1) 将每个高斯投影到图像平面上，计算2D协方差矩阵、投影半径 radii；
    // (2) 计算投影所占的tile块坐标和个数 tile tiles_touched；
    // (3) 如果用球谐系数，将其转换成RGB；
    // (4) 记录高斯的像素坐标 points_xy_image
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,      // 输出的 所有高斯 是否被裁剪的标志 数组，某位置为True表示：该高斯在当前相机的观测角度下，其RGB值3个的某个值 < 0，在后续渲染中不考虑它
		cov3D_precomp,          // 因预计算的3D协方差矩阵默认是空tensor，则传入的是一个 NULL指针
		colors_precomp,         // 因预计算的颜色默认是空tensor，则传入的是一个 NULL指针
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,              // 输出的 所有高斯 投影在当前相机图像平面的最大半径 数组
		geomState.means2D,  // 输出的 所有高斯 中心在当前相机图像平面的二维坐标 数组
		geomState.depths,   // 输出的 所有高斯 中心在当前相机坐标系下的z值 数组
		geomState.cov3D,    // 输出的 所有高斯 在世界坐标系下的3D协方差矩阵 数组
		geomState.rgb,      // 输出的 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组
		geomState.conic_opacity,    // 输出的 所有高斯 2D协方差的逆 和 不透明度 数组
		tile_grid,                  // CUDA网格的维度，grid.x是网格在x方向上的线程块数，grid.y是网格在y方向上的线程块数
		geomState.tiles_touched,    // 输出的 所有高斯 在当前相机图像平面覆盖的线程块 tile的个数 数组
		prefiltered                 // 预滤除的标志，默认为False
	), debug)

    //! 6. 高斯排序：根据高斯距离摄像机的远近来并行计算 每个高斯在 α-blending中的顺序
    // 在GPU上并行计算 每个高斯投影到当前相机图像平面上 2D高斯覆盖的 tile个数的 前缀和，结果存储在 point_offsets，提供了每个高斯覆盖tile区域的累加结束位置
    // 是为 所有高斯投影到图像平面上覆盖的所有 tile分配唯一的 ID
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space,  // 额外需要的临时显存空间
                                             geomState.scan_size,       // 临时显存空间的大小
                                             geomState.tiles_touched,   // 输入指针，已计算的 每个高斯 投影到当前相机图像平面覆盖的 tile个数的 数组
                                             geomState.point_offsets,   // 输出指针，指向一个数组，每个元素是 从第一个高斯到当前高斯所覆盖的所有 tile的 数量
                                             P      // 所有高斯的个数
                                             ), debug)

    // 计算所有高斯 投影到二维图像平面上覆盖的 tile的总个数 = 排序列表的长度
	int num_rendered;
    // 将 point_offsets数组的最后一个元素，即所有高斯投影到当前相机图像平面上所覆盖的 tile的 总数，从GPU复制到CPU的变量 num_rendered中
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

    // 为 BinningState分配显存，即每个高斯覆盖的 tile都有一个 BinningState数据，其存储着 排序前和排序后的 key、value列表
	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

    // 生成排序列表：对于每个要渲染的高斯，遍历其覆盖的tile，生成排序前的keys列表和values列表
    // point_list_keys_unsorted:    未排序的keys列表，分布顺序：大顺序：各高斯，小顺序：该高斯覆盖的各tile_ID。每个元素64bit，高32位存某高斯覆盖的tile_ID，低32位存某高斯中心在相机坐标系下的z（深度）值，即(tile_ID | 3D高斯的深度)
    // point_list_unsorted:         未排序的values列表，每个元素是 对应高斯的ID
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,  // 预处理计算的 所有高斯 中心在当前相机图像平面的二维坐标 数组
		geomState.depths,   // 预处理计算的 所有高斯 中心在当前相机坐标系下的z值（深度） 数组
		geomState.point_offsets,    // 所有高斯覆盖的 tile个数的 前缀和
		binningState.point_list_keys_unsorted,  // 输出的 未排序的 遍历所有高斯生成它们覆盖的tile的 keys 列表，每个元素是 (tile_ID | 3D高斯的深度)
		binningState.point_list_unsorted,       // 输出的 未排序的 遍历所有高斯生成它们覆盖的tile的 values 列表，每个元素是 对应高斯的ID
		radii,          // 预处理计算的 所有高斯 投影在当前相机图像平面的最大半径 数组
		tile_grid)      // CUDA网格的维度，grid.x是网格在x方向上的线程块数，grid.y是网格在y方向上的线程块数
	CHECK_CUDA(, debug)

    // 计算 tile总数的 二进制数中的 最高有效位的 位置，用于确定位操作的范围
	int bit = getHigherMsb(tile_grid.x * tile_grid.y);


    // 排序：对于每个tile，遍历落在其上的高斯，按各高斯的深度升序排序，生成排序后的keys列表和values列表
    // 按 key的大小，即tile_ID和3D高斯的深度，对 keys、values列表进行稳定的、并行、基数 升序排序
    // point_list_keys: 排序后的keys列表，分布顺序：大顺序：各tile_ID，小顺序：落在该tile内各高斯的深度
    // point_list:      排序后的values列表
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,    // 排序时用到的临时显存空间
		binningState.sorting_size,                     // 临时显存空间的大小
		binningState.point_list_keys_unsorted,  // 未排序的 每个高斯覆盖的所有 tile的 keys列表。 大顺序：各高斯，小顺序：该高斯覆盖的各tile_ID
        binningState.point_list_keys,           // 排序后的 keys列表。                       大顺序：各tile_ID，小顺序：落在该tile内各高斯的深度
		binningState.point_list_unsorted,   // 未排序的 每个高斯覆盖的所有 tile的 values列表。每个元素是[对应3D高斯的 ID]
        binningState.point_list,            // 排序后的 values列表
		num_rendered,   // 要排序的 tile总个数，即所有高斯 投影到二维图像平面上覆盖的 tile的总个数
        0,      // 指定时从最低位开始
        32 + bit    // 指定排序的最高位，表示排序的范围是从第 0位到第 32 + bit位。bit代表了 tile ID的最高位数。加上 32 是因为 tile ID和深度值分别占据了32位
        ), debug)

    // 将CUDA设备内存中的一块区域 imgState.ranges 数组中的所有元素初始化为 0（uint2是一个由两个 uint32_t组成的结构体，所以其大小是 8字节）
	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);


    // 根据排序后的keys列表，为每个 tile 计算 其在排序后的keys列表中的起始和终止位置，后续的渲染或处理步骤可以根据 tile ID 快速找到这个 tile 对应的高斯对象，而不需要再次进行复杂的查找或遍历
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,       // 排序的 tile总个数，即所有高斯 投影到二维图像平面上覆盖的 tile的总个数
			binningState.point_list_keys,   // 根据tile ID和高斯深度排序后的 keys列表
			imgState.ranges);   // 输出的 每个tile在 排序后的keys列表中的 起始和终止位置。索引：tile_ID；值[x,y)：该tile在keys列表中起始、终止位置，个数y-x：落在该tile_ID上的高斯的个数。也可以用[x,y)在排序后的values列表中索引到该tile触及的所有高斯ID
	CHECK_CUDA(, debug)

    // 如果传入的预计算的颜色 不是空指针，则是预计算的颜色
    //                    是空指针（默认），则是 preprocess中计算的 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;

    //! 7. 渲染: 在一个block上协作渲染一个tile内各像素的RGB颜色值，每个线程负责一个像素。具体实现在 forward.cu/renderCUDA
	CHECK_CUDA(FORWARD::render(
		tile_grid,     // 定义的CUDA网格的维度，grid.x是网格在x方向上的线程块数，grid.y是网格在y方向上的线程块数，(W/16，H/16)
        block,              // 定义的线程块 block的维度，(16, 16, 1)
		imgState.ranges,    // 每个tile在 排序后的keys列表中的 起始和终止位置。索引：tile ID，值[x,y)：该tile在keys列表中起始、终止位置，个数y-x：落在该tile_ID上的高斯的个数。也可以用[x,y)在排序后的values列表中索引到该tile触及的所有高斯ID
		binningState.point_list,    // 按 tile ID、高斯深度 排序后的 values列表，即 高斯ID 列表
		width, height,
		geomState.means2D,  // 已计算的 所有高斯 中心在当前相机图像平面的二维坐标 数组
		feature_ptr,        // 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组（每个高斯在当前观测方向下只有一个颜色，仅颜色强度分布不同）
		geomState.conic_opacity,    // 已计算的 所有高斯 2D协方差矩阵的逆 和 不透明度 数组
		imgState.accum_alpha,   // 输出的 渲染后每个像素 pixel的 累积的透射率 的数组
		imgState.n_contrib,     // 输出的 渲染每个像素 pixel穿过的高斯的个数，也是最后一个对渲染该像素RGB值 有贡献的高斯ID 的数组
		background,     // 背景颜色，默认为[1,1,1]，黑色
		out_color               // 输出的 RGB图像（加上了背景颜色）
        ), debug)

	return num_rendered;
}


/**
 * 反向传播，计算loss对前向传播输出tensor的 梯度
 */
void CudaRasterizer::Rasterizer::backward(
	const int P,    // 所有高斯的个数
    int D,  // 当前的球谐阶数
    int M,  // 每个高斯的球谐系数个数=16
    int R,  // 所有高斯覆盖的 tile的总个数
	const float* background,    // 背景颜色，默认为[1,1,1]，黑色
	const int width, int height,
	const float* means3D,   // 所有高斯 中心的世界坐标
	const float* shs,       // 所有高斯的 球谐系数，(N,16,3)
	const float* colors_precomp,    // python代码中 预计算的颜色，默认是空tensor
	const float* scales,    // 所有高斯的 缩放因子
	const float scale_modifier, // 缩放因子调节系数
	const float* rotations, // 所有高斯的 旋转四元数
	const float* cov3D_precomp, // python代码中 预计算的3D协方差矩阵，默认为空tensor
	const float* viewmatrix,    // 观测变换矩阵，W2C
	const float* projmatrix,    // 观测变换*投影变换矩阵，W2NDC = W2C * C2NDC
	const float* campos,    // 当前相机中心的世界坐标
	const float tan_fovx, float tan_fovy,
	const int* radii,   // 所有高斯 投影在当前相机图像平面上的最大半径 数组
	char* geom_buffer,  // 存储所有高斯 几何数据的 tensor：包括2D中心像素坐标、相机坐标系下的深度、3D协方差矩阵等
	char* binning_buffer,   // 存储所有高斯 排序数据的 tensor：包括未排序和排序后的 所有高斯覆盖的tile的 keys、values列表
	char* img_buffer,       // 存储所有高斯 渲染后数据的 tensor：包括累积的透射率、最后一个贡献的高斯ID
	const float* dL_dpix,   // 输入的 loss对渲染的RGB图像中每个像素颜色的 梯度（优化器输出的值，由优化器在训练迭代中自动计算）
	float* dL_dmean2D,  // 输出的 loss对所有高斯 中心投影到图像平面的像素坐标的 导数
	float* dL_dconic,   // 输出的 loss对所有高斯 2D协方差矩阵的 导数
	float* dL_dopacity, // 输出的 loss对所有高斯 不透明度的 导数
	float* dL_dcolor,   // 输出的 loss对所有高斯 在当前相机中心的观测方向下 的RGB颜色值 导数
	float* dL_dmean3D,  // 输出的 loss对所有高斯 中心世界坐标的 导数
	float* dL_dcov3D,   // 输出的 loss对所有高斯 3D协方差矩阵的 导数
	float* dL_dsh,      // 输出的 loss对所有高斯 球谐系数的 导数
	float* dL_dscale,   // 输出的 loss对所有高斯 缩放因子的 导数
	float* dL_drot,     // 输出的 loss对所有高斯 旋转四元数的 导数
	bool debug) // 默认为False
{
    // 这些缓冲区都是在前向传播的时候存下来的，现在拿出来用
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr) {
        // 如果传入的 所有高斯投影在当前相机图像平面上的最大半径数组为 空指针，则从geomState缓冲区中获取
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1); // 线程块 block（tile）的维度，(W/16, H/16, 1)
	const dim3 block(BLOCK_X, BLOCK_Y, 1);  // 一个block中 线程thread的维度，(16, 16, 1)

    // 如果传入的预计算的颜色 不是空指针，则是预计算的颜色
    //                    是空指针（默认），则是 preprocess中计算的 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;

    //! 反传中的 渲染 部分：计算 loss对 所有高斯的2D中心坐标、圆锥矩阵、不透明度和RGB等数据的 导数。具体实现在 backward.cu/renderCUDA
	CHECK_CUDA(BACKWARD::render(
		tile_grid,  // 线程块 block（tile）的维度，(W/16, H/16, 1)
		block,  // 一个block中 线程thread的维度，(16, 16, 1)
		imgState.ranges,    // 每个tile在 排序后的keys列表中的 起始和终止位置。索引：tile_ID；值[x,y)：该tile在keys列表中起始、终止位置，个数y-x：落在该tile_ID上的高斯的个数
		binningState.point_list,    // 按深度排序后的 所有高斯覆盖的tile的 values列表，每个元素是 对应高斯的ID
		width, height,
		background,     // 背景颜色，默认为[1,1,1]，黑色
		geomState.means2D,      // 所有高斯 中心投影在当前相机图像平面的二维坐标 数组
		geomState.conic_opacity,    // 所有高斯 2D协方差的逆 和 不透明度 数组
		color_ptr,      // 默认是 所有高斯 在当前相机中心的观测方向下 的RGB颜色值 数组
		imgState.accum_alpha,   // 渲染后每个像素 pixel的 累积的透射率 的数组
		imgState.n_contrib,     // 渲染每个像素 pixel穿过的高斯的个数，也是最后一个对渲染该像素RGB值 有贡献的高斯ID 的数组
		dL_dpix,    // 输入的 loss对渲染的RGB图像中每个像素颜色的 梯度（优化器输出的值，由优化器在训练迭代中自动计算）
		(float3*)dL_dmean2D,    // 输出的 loss对所有高斯 中心投影到图像平面的像素坐标的 导数
		(float4*)dL_dconic,     // 输出的 loss对所有高斯 2D协方差矩阵的 导数
		dL_dopacity,            // 输出的 loss对所有高斯 不透明度的 导数
		dL_dcolor),     // 输出的 loss对所有高斯 在当前相机中心的观测方向下 的RGB颜色值 导数
        debug)

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
    // 处理预处理的剩余部分
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;

    //! 反传中的 预处理 部分：
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot), debug)
}